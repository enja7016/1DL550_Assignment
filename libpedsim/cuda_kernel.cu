#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "heatmap_seq.h"

#include <stdio.h>

void kernel_addWithCuda(int *hm, int **heatmap, int size);

__global__ void kernel_add(int *d_hm, int **d_heatmap, int size)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;
	
	if (id < size) {
		// mul = SIZE*id
		// atomicAdd(&heatmap[id], hm);
		// atomicAdd(&heatmap[id], mul);
		d_heatmap[id] = d_hm + size*id;

	}
}

void kernel_setupHeatmap(int *hm, int *shm, int *bhm);
	kernel_addWithCuda(hm, heatmap, SIZE);
	kernel_addWithCuda(shm, scaled_heatmap, SCALED_SIZE);
	kernel_addWithCuda(bhm, blurred_heatmap, SCALED_SIZE);

// Helper function for using CUDA to add vectors in parallel.
void kernel_addWithCuda(int *hm, int **heatmap, int size)
{
	int *d_hm;
	int *d_heatmap;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		fprintf(stderr, "%s.\n", hipGetErrorString(hipGetLastError()));
		goto Error;
	}

	// Allocate GPU buffers for three vector
	cudaStatus = hipMalloc((void **) &d_hm, size*size*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void **) &d_heatmap, size*size*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vector from host memory to GPU buffers.
	cudaStatus = hipMemcpy(d_hm, hm, size*size*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_heatmap, heatmap, size*size*sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	kernel_add<<<1,size>>>(d_hm, d_heatmap, size);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}
	else
	{
		//fprintf(stderr, "Cuda launch succeeded! \n");
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(d_hm, hm, size*size*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(d_heatmap, heatmap, size*size*sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(d_hm);
	hipFree(d_heatmap);
	if (cudaStatus != 0){
		fprintf(stderr, "Cuda does not seem to be working properly.\n"); // This is not a good thing
	}
	else{
		fprintf(stderr, "Cuda functionality test succeeded.\n"); // This is a good thing
	}

	return cudaStatus;
}
