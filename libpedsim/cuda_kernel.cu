#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

/* ---------------------------
	SET HEATMAP FUNCTIONS
-----------------------------*/

void Ped::Model::setupHeatmapCuda()
{
	cout << "malloc";
	// Allocate memory on CPU
	int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
	int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

	heatmap = (int**)malloc(SIZE*sizeof(int*));
	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	// Initialize values, point to right memory
	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}


	int *desiredX = (int*)malloc(agents.size()*sizeof(int));
	int *desiredY = (int*)malloc(agents.size()*sizeof(int));

	for (int i = 0; i < agents.size(); i++)
	{
		Ped::Tagent* agent = agents[i];
		desiredX[i] = agent->getDesiredX();
		desiredY[i] = agent->getDesiredY();
	}




	hipMalloc(&d_desiredX, agents.size()*sizeof(int));
	hipMalloc(&d_desiredY, agents.size()*sizeof(int));
	// Allocate memory on GPU
	hipMalloc(&d_heatmap, SIZE*sizeof(int));
	hipMalloc(&d_scaled_heatmap, SCALED_SIZE*sizeof(int));
	hipMalloc(&d_blurred_heatmap, SCALED_SIZE*sizeof(int));

	// Copy memory from host to device
	hipMemcpy(d_heatmap, heatmap, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_scaled_heatmap, scaled_heatmap, SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_blurred_heatmap, blurred_heatmap, SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);
}

/* ---------------------------
	UPPDATE HEATMAP FUNCTIONS
  --------------------------*/ 

__global__ void kernel_fade(int *dev_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	// int y = blockIdx.y * blockDim.y + threadIdx.y;	
	if (x < SIZE)
	{
		dev_heatmap[x] = (int)round(dev_heatmap[x] * 0.80);
	}
}

__global__ void kernel_agents(int *dev_heatmap, int size_agents, int *desiredX, int *desiredY)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < size_agents){
		int x = desiredX[i];
		int y = desiredY[i];

		if(x>=0 && x<SIZE && y>=0 && y<SIZE)
			// intensify heat for better color results
			//&dev_heatmap[y][x] += 40;
			atomicAdd(&dev_heatmap[y*SIZE + x], 40);
	}
}

__global__ void kernel_clip(int *dev_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	if (x < SIZE ){
		dev_heatmap[x] = dev_heatmap[x] < 255 ? dev_heatmap[x] : 255;
	}
}

__global__ void kernel_scale(int *dev_heatmap, int *dev_scaled_heatmap)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
	int x = id / SIZE;
	int y = id % SIZE;
	if (id < SCALED_SIZE)
	{
		int value = dev_heatmap[id];
		for (int cellY = 0; cellY < CELLSIZE; cellY++)
		{
			for (int cellX = 0; cellX < CELLSIZE; cellX++)
			{
				int s_y = y * CELLSIZE + cellY;
                int s_x = x * CELLSIZE + cellX;
				dev_scaled_heatmap[s_y*SCALED_SIZE + s_x] = value;

			}
		}

	}
}

__global__ void kernel_blur(int *dev_heatmap, int *dev_blurred_heatmap, int *dev_scaled_heatmap)
{
	//weights for blur
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;	
	if (x >= 2 && x < SCALED_SIZE && y >= 2 && y < SCALED_SIZE)
	{
		int sum = 0;
		for (int k = -2; k < 3; k++)
		{
			for (int l = -2; l < 3; l++)
			{
				sum += w[2 + k][2 + l] * dev_scaled_heatmap[y + k][x + l];
			}
		}
		int value = sum / WEIGHTSUM;
		dev_blurred_heatmap[y][x] = 0x00FF0000 | value << 24;
	}
}


void Ped::Model::updateHeatmapCuda() 
{
	// Create streams
	hipStream_t stream1, stream2, stream3;
	hipStreamCreate(&stream1);
	hipStreamCreate(&stream2);
	hipStreamCreate(&stream3);

	hipEvent_t ev1;
	hipEventCreate(&ev1);
	// Create events

	// Fade heatmap
	kernel_fade<<<CELLSIZE, SIZE, 0, stream1>>>(d_heatmap);
	hipEventRecord(ev1, stream1);

	hipMemcpyAsync(d_desiredX, desiredX, agents.size()*sizeof(Ped::Tagent), hipMemcpyHostToDevice, stream2);
	hipMemcpyAsync(d_desiredY, desiredY, agents.size()*sizeof(Ped::Tagent), hipMemcpyHostToDevice, stream2);

	hipStreamWaitEvent(stream1, ev1);

	kernel_agents<<<1, agents.size(), 0, stream1>>>(d_heatmap, agents.size(), d_desiredX, d_desiredY);

	// Count how many agents want to go to each location

		// Count how many agents want to go to each location
	
	// for (int i = 0; i < agents.size(); i++)
	// {
		// Ped::Tagent* agent = agents[i];
		// int x = agent->getDesiredX();
		// int y = agent->getDesiredY();

		// if (x < 0 || x >= SIZE || y < 0 || y >= SIZE)
		// {
			// continue;
		// }
		// // intensify heat for better color results
		// d_heatmap[y][x] += 40;
	// }



	// int size_agents = agents.size();
	// Ped::Tagent *d_agents;
	// hipMalloc((void **)&d_agents, size_agents*sizeof(Ped::Tagent));
	// hipMemcpyAsync(d_agents, agents, size_agents*sizeof(Ped::Tagent), hipMemcpyHostToDevice, stream2);
	// hipStreamWaitEvent(stream1, ev1);
	// kernel_agents<<<1, size_agents, 0, stream1>>>(d_heatmap, d_agents, size_agents);
	// free(d_agents)
	// free(*d_agents)

	//Clip heatmap
	kernel_clip<<<1, SIZE, 0, stream1>>>(d_heatmap);

	//Scale heatmap
	kernel_scale<<<1, SIZE, 0, stream2>>>(d_heatmap, d_scaled_heatmap);

	// // Blur heatmap
	// kernel_blur<<<1, SIZE, 0, stream3>>>(d_heatmap, d_blurred_heatmap, d_scaled_heatmap);

}

int Ped::Model::getHeatmapSize() const {
	return SCALED_SIZE;
}

