#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

/* ---------------------------
	SET HEATMAP FUNCTIONS
-----------------------------*/

void Ped::Model::setupHeatmapCuda()
{
	// Allocate memory on CPU
	int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
	int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

	heatmap = (int**)malloc(SIZE*sizeof(int*));
	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	// Initialize values, point to right memory
	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}


	desiredX = new int[agents.size()];
	desiredY = new int[agents.size()];
	agentsSize = agents.size();
	agentsSizePtr = &agentsSize;

	//hipHostMalloc(&desiredX, agents.size()*sizeof(int));
	//hipHostMalloc(&desiredY, agents.size()*sizeof(int));
	
	for (int i = 0; i < agents.size(); i++)
	{
		desiredX[i] = agents[i]->getDesiredX();
		desiredY[i] = agents[i]->getDesiredY();
	}

	hipError_t status;
	status = hipStreamCreate(&s);
	if (status != hipSuccess) {
		fprintf(stderr, "stream fail\n");
	}

	hipMalloc(&d_desiredX, agents.size()*sizeof(int));
	hipMalloc(&d_desiredY, agents.size()*sizeof(int));

	hipMemcpy(d_desiredX, desiredX, agents.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_desiredY, desiredY, agents.size()*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_agentsSizePtr, sizeof(int*));
	hipMemcpy(d_agentsSizePtr, agentsSizePtr, sizeof(int*), hipMemcpyHostToDevice);


	// Allocate memory on GPU
	hipError_t cudaStatus;
	cudaStatus = hipMalloc(&d_heatmap, SIZE*SIZE*sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "heatmap memcpy failed!");
	}
	hipMemcpy(d_heatmap, heatmap[0], SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);
	// cout << heatmap[120][60] << "\n";

	hipMalloc(&d_scaled_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int));
	hipMemcpy(d_scaled_heatmap, scaled_heatmap[0], SCALED_SIZE*SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_blurred_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int*));
	hipMemcpy(d_blurred_heatmap, blurred_heatmap[0], SCALED_SIZE*SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);


	hipDeviceSynchronize();

	
	// Copy memory from host to device
}

/* ---------------------------
	UPPDATE HEATMAP FUNCTIONS
  --------------------------*/ 

__global__ void kernel_fade(int *dev_heatmap)
{

	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	//int y = blockIdx.y * blockDim.y + threadIdx.y;	
	// #if __CUDA_ARCH__ >= 200
	// {
	// 	printf("%d",x);
	// }
	// #endif
	if(x < SIZE) {
	dev_heatmap[x] = (int)round(dev_heatmap[x] * 0.80);

	}
}

__global__ void kernel_agents(int *dev_heatmap, int *size_agents, int *desiredX, int *desiredY)
{
	//int i = blockIdx.x * blockDim.x + threadIdx.x;
	int i = threadIdx.x;
	if(i < SIZE*SIZE){
		int x = desiredX[i];
		int y = desiredY[i];
		atomicAdd(&dev_heatmap[y*SIZE + i], 40);
		//#if __CUDA_ARCH__ >= 200
		//{
		//	printf("AGENTS: %d \n",dev_heatmap[y*SIZE+x]);
		//}
		//#endif
	}
}

__global__ void kernel_clip(int *dev_heatmap, int *size_agents, int *desiredX, int *desiredY)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;	
	if (i < SIZE*SIZE){
		// int x = desiredX[i];
		// int y = desiredY[i];
		// atomicMin(&dev_heatmap[i], 255);
		dev_heatmap[i] = dev_heatmap[i] < 255 ? dev_heatmap[i] : 255;
		// #if __CUDA_ARCH__ >= 200
		// {
		// 	printf("CLIP: %d \n",dev_heatmap[i]);
		// }
		// #endif
	}
}

__global__ void kernel_scale(int *dev_heatmap, int *dev_scaled_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.x * blockDim.x + threadIdx.x;	
	if(x < SIZE*SIZE) {
		int value = dev_heatmap[y*SIZE + x];
		for (int cellY = 0; cellY < CELLSIZE; cellY++)
		{
			for (int cellX = 0; cellX < CELLSIZE; cellX++)
			{
				dev_scaled_heatmap[(y*CELLSIZE+cellY)*SIZE*CELLSIZE + x*CELLSIZE*cellX] = value;
			}
		}
	}
}

__global__ void kernel_blur(int *dev_heatmap, int *dev_blurred_heatmap, int *dev_scaled_heatmap)
{
	//weights for blur
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;	
	if (x >= 2 && x < SCALED_SIZE && y >= 2 && y < SCALED_SIZE)
	{
		int sum = 0;
		for (int k = -2; k < 3; k++)
		{
			for (int l = -2; l < 3; l++)
			{
				sum += w[2 + k][2 + l] * dev_scaled_heatmap[(y + k) * SCALED_SIZE + (x + l)];
			}
		}
		int value = sum / WEIGHTSUM;
		dev_blurred_heatmap[y*SCALED_SIZE + x] = 0x00FF0000 | value << 24;
	}
	#if __CUDA_ARCH__ >= 200
	{
		printf("%d",dev_blurred_heatmap[0]);
	}
	#endif
}


void Ped::Model::updateHeatmapCuda() 
{

	for (int i = 0; i < agents.size(); i++)
	{
		desiredX[i] = agents[i]->getDesiredX();
		desiredY[i] = agents[i]->getDesiredY();
	}

	hipMemcpyAsync(d_desiredX, desiredX, agents.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpyAsync(d_desiredY, desiredY, agents.size()*sizeof(int), hipMemcpyHostToDevice);

	//cout << d_desiredX[0] << "\n";

	// dim3 threads_per_block(32, 32);
    // dim3 num_blocks(SIZE / threads_per_block.x, SIZE / threads_per_block.y);
	// Fade heatmap
	kernel_fade<<<SIZE, SIZE>>>(d_heatmap);

	//heatmapFading<<<SIZE, SIZE, 0, stream>>>(heatmap_tmp);

    // int threads_per_blocki = 1024;
    // int num_blocksi = (agents.o1337@rackham3 1DL550_Assignment]$ interactive -A uppmax2023-2-4 -M snowy -p core -n 1 -c 4 -t 30:00


	kernel_agents<<<SIZE, SIZE>>>(d_heatmap, d_agentsSizePtr, d_desiredX, d_desiredY);

	hipError_t cudaStatus;

	cudaStatus= hipMemcpyAsync(d_heatmap[0], heatmap, SIZE*sizeof(int*), hipMemcpyDeviceToHost);
	if(cudaStatus != hipSuccess) {
		fprintf(stderr, "FUCK OFF\n");
	}


	//cout << "NIKLAS: "<< heatmap[500][500] << "\n";


	// int size_agents = agents.size();
	// Ped::Tagent *d_agents;
	// hipMalloc((void **)&d_agents, size_agents*sizeof(Ped::Tagent));
	// hipMemcpyAsync(d_agents, agents, size_agents*sizeof(Ped::Tagent), hipMemcpyHostToDevice, stream2);
	// hipStreamWaitEvent(stream1, ev1);
	// kernel_agents<<<1, size_agents, 0, stream1>>>(d_heatmap, d_agents, size_agents);
	// free(d_agents)
	// free(*d_agents)

	 

	//Clip heatmap
	kernel_clip<<<SIZE, SIZE>>>(d_heatmap, d_agentsSizePtr, d_desiredX, d_desiredY);


	//Scale heatmap
	// kernel_scale<<<SIZE, SIZE>>>(d_heatmap, d_scaled_heatmap);

	// // Blur heatmap
	// kernel_blur<<<SIZE,SIZE>>>(d_heatmap, d_blurred_heatmap, d_scaled_heatmap);


	hipMemcpy(blurred_heatmap, d_blurred_heatmap, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
	// cout << blurred_heatmap[0][0] << "\n";

}

int Ped::Model::getHeatmapSize() const {
	return SCALED_SIZE;
}

