#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>

/* ---------------------------
	SET HEATMAP FUNCTIONS
-----------------------------*/

void Ped::Model::setupHeatmapCuda()
{
	// Allocate memory on CPU
	int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
	int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

	heatmap = (int**)malloc(SIZE*sizeof(int*));
	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	// Initialize values, point to right memory
	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}


	desiredX = (int*)malloc(agents.size()*sizeof(int));
	desiredY = (int*)malloc(agents.size()*sizeof(int));
	//hipHostMalloc(&desiredX, agents.size()*sizeof(int));
	//hipHostMalloc(&desiredY, agents.size()*sizeof(int));
	
	hipMalloc(&d_desiredX, agents.size()*sizeof(int));
	hipMalloc(&d_desiredY, agents.size()*sizeof(int));



	// Allocate memory on GPU
	hipMalloc(&d_heatmap, SIZE*sizeof(int));
	hipMalloc(&d_scaled_heatmap, SCALED_SIZE*sizeof(int));
	hipMalloc(&d_blurred_heatmap, SCALED_SIZE*sizeof(int));

	// Copy memory from host to device
	hipMemcpy(d_heatmap, heatmap, SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_scaled_heatmap, scaled_heatmap, SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_blurred_heatmap, blurred_heatmap, SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);
}

/* ---------------------------
	UPPDATE HEATMAP FUNCTIONS
  --------------------------*/ 

__global__ void kernel_fade(int *dev_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;	
	dev_heatmap[y*SIZE + x] = (int)round(dev_heatmap[y*SIZE + x] * 0.80);
}

__global__ void kernel_agents(int *dev_heatmap, int size_agents, int *desiredX, int *desiredY)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < size_agents){
		int x = (int) desiredX[i];
		int y = (int) desiredY[i];
		atomicAdd(&dev_heatmap[y*SIZE + x], 40);

			// intensify heat for better color results
			//&dev_heatmap[y][x] += 40;
	}
}

__global__ void kernel_clip(int *dev_heatmap, int size_agents, int *desiredX, int *desiredY)
{
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	if (tid < size_agents) {
		int x = (int) desiredX[tid];
		int y = (int) desiredY[tid];

		atomicMin(&dev_heatmap[y*SIZE + x], 255);
	}
}


__global__ void kernel_scale(int *dev_heatmap, int *dev_scaled_heatmap)
{
	int ytid = blockIdx.y * blockDim.y + threadIdx.y;	
	int xtid = blockIdx.x * blockDim.x + threadIdx.x;	
	int value = dev_heatmap[ytid*SIZE + xtid];
	for (int cellY = 0; cellY < CELLSIZE; cellY++)
	{
		for (int cellX = 0; cellX < CELLSIZE; cellX++)
		{
			dev_scaled_heatmap[(ytid*CELLSIZE+cellY) * SIZE * CELLSIZE + xtid*CELLSIZE*cellX] = value;

		}

	}
}

__global__ void kernel_blur(int *dev_heatmap, int *dev_blurred_heatmap, int *dev_scaled_heatmap)
{
	const int w[5][5] = 
	{{1, 4, 7, 4, 1},
	{4, 16, 26, 16, 4},
	{7, 26, 41, 26, 7},
	{4, 16, 26, 16, 4},
	{1, 4, 7, 4, 1}};


	int y = blockIdx.y * blockDim.y + threadIdx.y;	
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	if (x >= 2 && x < SCALED_SIZE && y >= 2 && y < SCALED_SIZE)
	{
		int sum = 0;
		for (int k = -2; k < 3; k++)
		{
			for (int l = -2; l < 3; l++)
			{
				sum += w[2 + k][2 + l] * dev_scaled_heatmap[(y + k) * SCALED_SIZE + x + l];
			}
		}
		int value = sum / WEIGHTSUM;
		#if __CUDA_ARCH__ >= 200
			printf("SCALING ERROR: %i", value);
		#endif
		dev_blurred_heatmap[y*SCALED_SIZE + x] = 0x00FF0000 | value << 24;
	}
}


void Ped::Model::updateHeatmapCuda() 
{

	hipMemcpy(d_desiredX, desiredX, agents.size() * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_desiredY, desiredY, agents.size() * sizeof(int), hipMemcpyHostToDevice);
	//cout << d_desiredX[0] << "\n";

	dim3 threads_per_block(32, 32);
    dim3 num_blocks(SIZE / threads_per_block.x, SIZE / threads_per_block.y);
	// Fade heatmap
	kernel_fade<<<num_blocks, threads_per_block>>>(d_heatmap);

    int threads_per_blocki = 1024;
    int num_blocksi = (agents.size() + threads_per_blocki - 1) / threads_per_blocki;
	kernel_agents<<<num_blocksi, threads_per_blocki>>>(d_heatmap, agents.size(), d_desiredX, d_desiredY);


	//Clip heatmap
	kernel_clip<<<num_blocks, threads_per_block>>>(d_heatmap, agents.size(), d_desiredX, d_desiredY);

	//Scale heatmap
	kernel_scale<<<num_blocks, threads_per_block>>>(d_heatmap, d_scaled_heatmap);

	// Blur heatmap
	dim3 num_blocks_SCALED(SCALED_SIZE / threads_per_block.x, SCALED_SIZE / threads_per_block.y);
	kernel_blur<<<num_blocks_SCALED,threads_per_block >>>(d_heatmap, d_blurred_heatmap, d_scaled_heatmap);

	hipMemcpy(blurred_heatmap, d_blurred_heatmap, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);
}

int Ped::Model::getHeatmapSize() const {
	return SCALED_SIZE;
}

