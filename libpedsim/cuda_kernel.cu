#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include "ped_model.h"

#include <cstdlib>
#include <iostream>
#include <cmath>
using namespace std;

// Memory leak check with msvc++
#include <stdlib.h>
#define SIZE 1024

/* ---------------------------
	SET HEATMAP FUNCTIONS
-----------------------------*/

void Ped::Model::setupHeatmapCuda()
{
	cout << "malloc";
	// Allocate memory on CPU
	int *hm = (int*)calloc(SIZE*SIZE, sizeof(int));
	int *shm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));
	int *bhm = (int*)malloc(SCALED_SIZE*SCALED_SIZE*sizeof(int));

	heatmap = (int**)malloc(SIZE*sizeof(int*));
	scaled_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));
	blurred_heatmap = (int**)malloc(SCALED_SIZE*sizeof(int*));

	// Initialize values, point to right memory
	for (int i = 0; i < SIZE; i++)
	{
		heatmap[i] = hm + SIZE*i;
	}
	for (int i = 0; i < SCALED_SIZE; i++)
	{
		scaled_heatmap[i] = shm + SCALED_SIZE*i;
		blurred_heatmap[i] = bhm + SCALED_SIZE*i;
	}


	desiredX = new int[agents.size()];
	desiredY = new int[agents.size()];
	agents = agents.size();
	agentsSize = &agents;

	//hipHostMalloc(&desiredX, agents.size()*sizeof(int));
	//hipHostMalloc(&desiredY, agents.size()*sizeof(int));
	
	for (int i = 0; i < agents.size(); i++)
	{
		desiredX[i] = agents[i]->getDesiredX();
		desiredY[i] = agents[i]->getDesiredY();
	}

	hipMalloc(&d_desiredX, agents.size()*sizeof(int));
	hipMalloc(&d_desiredY, agents.size()*sizeof(int));

	hipMemcpy(d_desiredX, desiredX, agents.size()*sizeof(int));
	hipMemcpy(d_desiredY, desiredY, agents.size()*sizeof(int));

	hipMalloc(&d_agentsSize, sizeof(int*));
	cudaMempy(d_agentsSize, agentsSize, sizeof(int*), hipMemcpyHostToDevice);


	// Allocate memory on GPU
	hipMalloc(&d_heatmap, SIZE*SIZE*sizeof(int));
	hipMemcpy(d_heatmap, heatmap[0], SIZE*SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_scaled_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int));
	hipMemcpy(d_scaled_heatmap, scaled_heatmap[0], SCALED_SIZE*SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);

	hipMalloc(&d_blurred_heatmap, SCALED_SIZE*SCALED_SIZE*sizeof(int*));
	hipMemcpy(d_blurred_heatmap, blurred_heatmap[0], SCALED_SIZE*SCALED_SIZE*sizeof(int), hipMemcpyHostToDevice);
	hipDeviceSynchronize();

	
	// Copy memory from host to device
}

/* ---------------------------
	UPPDATE HEATMAP FUNCTIONS
  --------------------------*/ 

__global__ void kernel_fade(int *dev_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	// int y = blockIdx.y * blockDim.y + threadIdx.y;	
	if (x < SIZE)
	{
		dev_heatmap[x] = (int)round(dev_heatmap[x] * 0.80);
	}
}

__global__ void kernel_agents(int *dev_heatmap, int size_agents, int *desiredX, int *desiredY)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if(i < size_agents){
		int x = desiredX[i];
		int y = desiredY[i];

		if(x>=0 && x<SIZE && y>=0 && y<SIZE)
			// intensify heat for better color results
			//&dev_heatmap[y][x] += 40;
			atomicAdd(&dev_heatmap[y*SIZE + x], 40);
	}
}

__global__ void kernel_clip(int *dev_heatmap)
{
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	if (x < SIZE ){
		dev_heatmap[x] = dev_heatmap[x] < 255 ? dev_heatmap[x] : 255;
	}
}

__global__ void kernel_scale(int *dev_heatmap, int *dev_scaled_heatmap)
{
	int id = blockIdx.x * blockDim.x + threadIdx.x;	
	int x = id / SIZE;
	int y = id % SIZE;
	if (id < SCALED_SIZE)
	{
		int value = dev_heatmap[id];
		for (int cellY = 0; cellY < CELLSIZE; cellY++)
		{
			for (int cellX = 0; cellX < CELLSIZE; cellX++)
			{
				int s_y = y * CELLSIZE + cellY;
                int s_x = x * CELLSIZE + cellX;
				dev_scaled_heatmap[s_y*SCALED_SIZE + s_x] = value;

			}
		}

	}
}

__global__ void kernel_blur(int *dev_heatmap, int *dev_blurred_heatmap, int *dev_scaled_heatmap)
{
	//weights for blur
	const int w[5][5] = {
		{ 1, 4, 7, 4, 1 },
		{ 4, 16, 26, 16, 4 },
		{ 7, 26, 41, 26, 7 },
		{ 4, 16, 26, 16, 4 },
		{ 1, 4, 7, 4, 1 }
	};
	int x = blockIdx.x * blockDim.x + threadIdx.x;	
	int y = blockIdx.y * blockDim.y + threadIdx.y;	
	if (x >= 2 && x < SCALED_SIZE && y >= 2 && y < SCALED_SIZE)
	{
		int sum = 0;
		for (int k = -2; k < 3; k++)
		{
			for (int l = -2; l < 3; l++)
			{
				sum += w[2 + k][2 + l] * dev_scaled_heatmap[y + k][x + l];
			}
		}
		int value = sum / WEIGHTSUM;
		dev_blurred_heatmap[y][x] = 0x00FF0000 | value << 24;
	}
}


void Ped::Model::updateHeatmapCuda() 
{



	for (int i = 0; i < agents.size(); i++)
	{
		desiredX[i] = agents[i]->getDesiredX();
		desiredY[i] = agents[i]->getDesiredY();
	}


	hipMemcpy(d_desiredX, desiredX, agents.size()*sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(d_desiredY, desiredY, agents.size()*sizeof(int), hipMemcpyHostToDevice);

	//cout << d_desiredX[0] << "\n";

	// dim3 threads_per_block(32, 32);
    // dim3 num_blocks(SIZE / threads_per_block.x, SIZE / threads_per_block.y);
	// Fade heatmap
	kernel_fade<<<SIZE, SIZE>>>(d_heatmap);
	//heatmapFading<<<SIZE, SIZE, 0, stream>>>(heatmap_tmp);

    // int threads_per_blocki = 1024;
    // int num_blocksi = (agents.size() + threads_per_blocki - 1) / threads_per_blocki;
	kernel_agents<<<1, agents.size()>>>(d_heatmap, d_agentsSize, d_desiredX, d_desiredY);


	// int size_agents = agents.size();
	// Ped::Tagent *d_agents;
	// hipMalloc((void **)&d_agents, size_agents*sizeof(Ped::Tagent));
	// hipMemcpyAsync(d_agents, agents, size_agents*sizeof(Ped::Tagent), hipMemcpyHostToDevice, stream2);
	// hipStreamWaitEvent(stream1, ev1);
	// kernel_agents<<<1, size_agents, 0, stream1>>>(d_heatmap, d_agents, size_agents);
	// free(d_agents)
	// free(*d_agents)

	//Clip heatmap
	kernel_clip<<<SIZE, SIZE>>>(d_heatmap, agents.size(), d_desiredX, d_desiredY);

	//Scale heatmap
	kernel_scale<<<SIZE, SIZE>>>(d_heatmap, d_scaled_heatmap);

	// Blur heatmap
	kernel_blur<<<SIZE,SIZE >>>(d_heatmap, d_blurred_heatmap, d_scaled_heatmap);

	hipMemcpy(blurred_heatmap, d_blurred_heatmap, SCALED_SIZE * SCALED_SIZE * sizeof(int), hipMemcpyDeviceToHost);

}

int Ped::Model::getHeatmapSize() const {
	return SCALED_SIZE;
}

